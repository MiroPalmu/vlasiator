#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "open_acc_map_h.cuh"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

/*
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ));
static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}
*/
__global__ void cudaFunction(int *b)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  printf("CUDA [%d]: \n", index);
  if(index<CUDASIZE)
  {
    b[index] = b[index]+index;
  }
}

void wrapper(int c)
{
  printf("STAGE 3\n");
  printf("c: %d\n", c);
  int b[CUDASIZE];
  int *dev_b;
  //HANDLE_ERROR( hipMalloc((void**)&dev_b, CUDASIZE * sizeof(int)) );
  hipMalloc((void**)&dev_b, CUDASIZE * sizeof(int));
  for(int a_c=0; a_c<CUDASIZE; a_c++)
  {
    b[a_c] = c-a_c;
  }
  printf("before: b: %d\n", b[0]);
	hipMemcpy(dev_b, b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  cudaFunction<<<BLOCKS, THREADS>>>(dev_b);
  hipMemcpy(b, dev_b, CUDASIZE*sizeof(int), hipMemcpyHostToDevice);
  printf("after: b: %d\n", b[CUDASIZE-1]);
  hipFree(dev_b);
}
